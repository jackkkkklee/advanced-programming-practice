/*
Author: Congwang Li
Class:  ECE6122
Last Date Modified: 8/11/2022
Description:
    calculate temperature in thin pale by cuda
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <getopt.h>
#include <iostream>
#include <fstream>
#include <chrono>
/*!
 *
 * @param g a matrix contains new value
 * @param h a matrix contains temperature
 * @param interior size of interior
 */
__global__ void heatCalculation(double * g, double * h, const int n)
{
    //get row based address to
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    //make sure extra threads do not do any work
    if (i > 0 && i < n -1  && j > 0 && j < n -1 )
    {
        h[i + j * n] = 0.25 * (g[(i + 1) + j * n] + g[(i - 1) + j * n] + g[i + (j + 1) * n] + g[i + (j - 1) * n]);
    }
}
/*!
 * init temperature
 * @param matrix a matrix to be init by base temperature
 * @param n number of points
 */
void initHeatMatrix(double * matrix, const int n)
{
    //set all interior points to 20
    for (int j = 0; j < n; j++)
    {
        for (int i = 0; i < n; i++)
        {
            matrix[i + n * j] = 20.0;
        }

    }

    //set some top part to 100
    for (int i = 0; i < n; i++)
    {
        if (i > (int)(0.3 * n) && i < (int)(0.7 * n))
        {
            matrix[i] = 100.0;
        }
    }

}
int main(int argc, char* args[]) {
    int n = 0;
    int I = 0;
    int opt;
    //parse args
    while ((opt = getopt(argc, args, "n:I:")) != -1)
    {
        if (opt == 'n')
        {
            n = atoi(optarg) + 2;
        }
        if (opt == 'I')
        {
            I = atoi(optarg);
        }
    }
    if (n <= 0 || I <= 0)
    {
        std::cout << "invalid input!" << std::endl;
        return 0;
    }

    //tow matrix to contain temperature
    double* heatMatrixG;
    double* heatMatrixH;

    //allocate unified memory
    hipMallocManaged(&heatMatrixG, n * n * sizeof(double));
    hipMallocManaged(&heatMatrixH, n * n * sizeof(double));

    // set dimBLock and grid size
    dim3 dimBlock(32, 32);
    dim3 dimGrid(ceil((n+31) / (32)) , ceil((n+31) / (32) ));

    //init heat matrix
    initHeatMatrix(heatMatrixH,n);
    initHeatMatrix(heatMatrixG,n);//todo maybe copy could be faster

    //calculate time

    auto startTime = std::chrono::system_clock::now();

    // run I times
    for (int i = 0; i < I; i+=2)
    {
        //change the param order of g and h  to avoid copy
        heatCalculation<<<dimGrid, dimBlock>>>(heatMatrixG, heatMatrixH, n);
        heatCalculation<<<dimGrid, dimBlock>>>(heatMatrixH, heatMatrixG, n);
    }
    //the result matrix determined by I since it is calculated twice in one for cycle
    double* res = heatMatrixG;
    if (I % 2 != 0)
    {
        res = heatMatrixH;
    }

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    auto endTime = std::chrono::system_clock::now();
    printf("Thin plate calculation took %.3f milliseconds",(float )std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count()/1000);

    //output to file
    std::ofstream resultFile;
    resultFile.open("finalTemperatures.csv");

    resultFile.precision(4);

    for (int j = 0; j < n; j++)
    {
        for (int i = 0; i < n; i++)
        {
            resultFile << res[i + j * n] << "," << std::fixed;
        }
        resultFile << std::endl;
    }
    resultFile.close();

    //free
    hipFree(heatMatrixG);
    hipFree(heatMatrixH);
    return 0;
}
